#include <iostream>
#include <iomanip>
#include <vector>
#include <complex>
#include <hip/hip_runtime.h>
#include <cstdint>

#include <hipfft/hipfft.h>

#define BLOCKSIZE 256
#define NAIVE_SIZE 2048

__global__ void subsetSumKernelDp(uint32_t *w, uint32_t *global_dp, const int n, uint32_t sum, uint32_t *blocks) {
    uint32_t T = sum+1;

    uint32_t currentIdx = blockIdx.x * (T);
    uint32_t previousIdx = (1 - blockIdx.x % 2) * T;


    uint32_t interval = (T+BLOCKSIZE - 1)/BLOCKSIZE;
    uint32_t start_index = threadIdx.x*interval;
    uint32_t end_index = start_index+interval+1;
    end_index = (end_index < (T+1)) ? end_index : (T+1);
    

    uint32_t l = NAIVE_SIZE * blockIdx.x;
    uint32_t r = l + NAIVE_SIZE;
    r = (r < n) ? r : (n);
    if (threadIdx.x == 0){
        global_dp[previousIdx]=1;
    }
    __syncthreads();
    for (uint32_t i = l; i < r; i++) {
        const uint32_t x = w[i];

        __syncthreads(); // Synchronize before starting to read/write

        for (uint32_t dp_block = start_index; dp_block < end_index; dp_block++) {
            // Compute dp_current based on dp_previous
            if (dp_block >= x) {
                global_dp[currentIdx + dp_block] = global_dp[previousIdx + dp_block] || global_dp[previousIdx + dp_block - x];
            } else {
                global_dp[currentIdx + dp_block] = global_dp[previousIdx + dp_block];
            }
        }

        __syncthreads(); // Synchronize after writing to global memory

        // Swap logic: switch currentIdx and previousIdx for the next iteration
        if (i < (r-1)) {
            // Swap the indices
            uint32_t temp = currentIdx;
            currentIdx = previousIdx;
            previousIdx = temp;
        }

        __syncthreads(); // Synchronize before starting the next iteration

        
    }

    for (uint32_t dp_block = start_index; dp_block < end_index; dp_block++) {
        uint32_t flat_idx = blockIdx.x * (T + 1) + threadIdx.x+dp_block;
        blocks[flat_idx] = global_dp[currentIdx + dp_block];
    }
}

__global__ void pointwiseMultiply(hipfftDoubleComplex *input1, 
                                  hipfftDoubleComplex *input2, 
                                  hipfftDoubleComplex *result, 
                                  int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < n) {
        hipfftDoubleComplex a = input1[index];
        hipfftDoubleComplex b = input2[index];
        result[index] = make_hipDoubleComplex(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
    }
}

std::vector<std::complex<double>> fftConvolutionCuFFT(const std::vector<std::complex<double>>& input1,
                         const std::vector<std::complex<double>>& input2) {
    int n = input1.size(); // Assuming input1 and input2 are the same size
    std::vector<std::complex<double>> result(n);

    // Allocate memory on the device
    hipfftDoubleComplex *d_input1, *d_input2, *d_result;
    hipMalloc(&d_input1, n * sizeof(hipfftDoubleComplex));
    hipMalloc(&d_input2, n * sizeof(hipfftDoubleComplex));
    hipMalloc(&d_result, n * sizeof(hipfftDoubleComplex));

    // Copy host data to device
    hipMemcpy(d_input1, input1.data(), n * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_input2, input2.data(), n * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);

    // Create a cuFFT plan
    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_Z2Z, 1);

    // Execute forward FFT
    hipfftExecZ2Z(plan, d_input1, d_input1, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan, d_input2, d_input2, HIPFFT_FORWARD);

    // Perform point-wise multiplication
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    pointwiseMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_input1, d_input2, d_result, n);

    // Execute inverse FFT
    hipfftExecZ2Z(plan, d_result, d_result, HIPFFT_BACKWARD);

    // Copy result back to host
    hipMemcpy(result.data(), d_result, n * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);

    // Clean up
    hipfftDestroy(plan);
    hipFree(d_input1);
    hipFree(d_input2);
    hipFree(d_result);
    return result;
}

std::vector<std::vector<std::complex<double>>> convertToComplex2D(
    const std::vector<uint32_t>& input, 
    uint32_t num_blocks, 
    uint32_t T) {

    std::vector<std::vector<std::complex<double>>> output(num_blocks, std::vector<std::complex<double>>(T));
    
    for (uint32_t i = 0; i < num_blocks; ++i) {
        for (uint32_t j = 0; j < T; ++j) {
            // Convert input[i * T + j] to std::complex<double>
            double realPart = static_cast<double>(input[i * T + j]);
            output[i][j] = std::complex<double>(realPart, 0.0); // imaginary part is 0
        }
    }

    return output;
}

bool solve_fft(const std::vector<uint32_t>& w, const uint32_t T){
    const int n = std::size(w);
    uint32_t num_blocks = (n + NAIVE_SIZE - 1) / NAIVE_SIZE;
    //std::cout << "numblocks " << num_blocks << "\n";
    const int num_iterations = std::__lg(num_blocks);
    

    size_t global_dp_size = 2 * (T+1) * num_blocks * sizeof(uint32_t);
    uint32_t* global_dp;
    
    
    // blocks of size num_blocks*sum+1
    std::vector<uint32_t> blocks(num_blocks * (T + 1), 0);
    hipMalloc(&global_dp, global_dp_size);
    
    uint32_t* d_w;
    hipMalloc(&d_w, n * sizeof(uint32_t));
    hipMemcpy(d_w, w.data(), n * sizeof(uint32_t), hipMemcpyHostToDevice);

    

    uint32_t* d_blocks;
    hipMalloc(&d_blocks, num_blocks*(T+1)*sizeof(uint32_t));

    
    subsetSumKernelDp<<<num_blocks, BLOCKSIZE>>>(d_w, global_dp, n, T, d_blocks);
    
    hipMemcpy(blocks.data(), d_blocks, num_blocks*(T+1) * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipFree(d_blocks);
    hipFree(global_dp);
    hipFree(d_w);
    
    // create an array of cuda complex doubles
    std::vector fft_outputs(num_iterations + 1, std::vector<std::vector<std::complex<double>>>(num_blocks));
    fft_outputs[0] = convertToComplex2D(blocks, num_blocks, T+1);
    //std::cout << fft_outputs[0][0][5];
    
    for (int iter = 0, iter_num_blocks = num_blocks; iter < num_iterations; iter++, iter_num_blocks = (iter_num_blocks + 1) / 2) {
        const int next_iter_num_blocks = (iter_num_blocks + 1) / 2;

        for (int i = 0; i < next_iter_num_blocks; i++) {
            if (2 * i + 1 < iter_num_blocks) {
                fft_outputs[iter + 1][i] = fftConvolutionCuFFT(fft_outputs[iter][2 * i], fft_outputs[iter][2 * i + 1]);

            } else {
                fft_outputs[iter + 1][i] = std::move(fft_outputs[iter][2 * i]);
            }
        }
    }

    

    //std::cout << "Time spent combining subarray solutions: " << std::fixed << std::setprecision(10) << (convolution_timer.get_duration<std::chrono::microseconds>() / 1e6) << '\n';

    bool is_possible = (fft_outputs[num_iterations][0][T].real()>0);

    
    
    return is_possible;

}
