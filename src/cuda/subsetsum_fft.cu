#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <complex>
#include <hip/hip_runtime.h>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "timer.h"

#define BLOCKSIZE 512
#define NAIVE_SIZE 1024

__global__ void subsetSumKernelDp(uint32_t *w, uint32_t *global_dp, const int n, uint32_t sum, uint32_t *blocks) {
    uint32_t T = sum+1;

    uint32_t currentIdx = blockIdx.x * (T);
    uint32_t previousIdx = (1 - blockIdx.x % 2) * T;


    uint32_t interval = (T+BLOCKSIZE - 1)/BLOCKSIZE;
    uint32_t start_index = threadIdx.x*interval;
    uint32_t end_index = start_index+interval+1;
    end_index = (end_index < (T+1)) ? end_index : (T+1);
    

    uint32_t l = NAIVE_SIZE * blockIdx.x;
    uint32_t r = l + NAIVE_SIZE;
    r = (r < n) ? r : (n);
    if (threadIdx.x == 0){
        global_dp[previousIdx]=1;
    }
    __syncthreads();
    for (uint32_t i = l; i < r; i++) {
        const uint32_t x = w[i];

        __syncthreads();

        for (uint32_t dp_block = start_index; dp_block < end_index; dp_block++) {

            if (dp_block >= x) {
                global_dp[currentIdx + dp_block] = global_dp[previousIdx + dp_block] || global_dp[previousIdx + dp_block - x];
            } else {
                global_dp[currentIdx + dp_block] = global_dp[previousIdx + dp_block];
            }
        }

        __syncthreads();

        if (i < (r-1)) {

            uint32_t temp = currentIdx;
            currentIdx = previousIdx;
            previousIdx = temp;
        }

        __syncthreads(); 

        
    }

    for (uint32_t dp_block = start_index; dp_block < end_index; dp_block++) {
        uint32_t flat_idx = blockIdx.x * (T + 1) + threadIdx.x+dp_block;
        blocks[flat_idx] = global_dp[currentIdx + dp_block];
    }
}

__global__ void pointwiseMultiply(hipfftDoubleComplex *input1, 
                                  hipfftDoubleComplex *input2, 
                                  hipfftDoubleComplex *result, 
                                  int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < n) {
        hipfftDoubleComplex a = input1[index];
        hipfftDoubleComplex b = input2[index];
        result[index] = make_hipDoubleComplex(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
    }
}

std::vector<std::complex<double>> fftConvolutionCuFFT(const std::vector<std::complex<double>>& input1,
                         const std::vector<std::complex<double>>& input2) {
    int n = input1.size(); 
    std::vector<std::complex<double>> result(n);
    Timer memory_timer;
    memory_timer.start();

    hipfftDoubleComplex *d_input1, *d_input2, *d_result;
    hipMalloc(&d_input1, n * sizeof(hipfftDoubleComplex));
    hipMalloc(&d_input2, n * sizeof(hipfftDoubleComplex));
    hipMalloc(&d_result, n * sizeof(hipfftDoubleComplex));


    hipMemcpy(d_input1, input1.data(), n * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_input2, input2.data(), n * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
    memory_timer.end();
    Timer actual_ops;
    actual_ops.start();

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_Z2Z, 1);

    hipfftExecZ2Z(plan, d_input1, d_input1, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan, d_input2, d_input2, HIPFFT_FORWARD);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    pointwiseMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_input1, d_input2, d_result, n);

    hipfftExecZ2Z(plan, d_result, d_result, HIPFFT_BACKWARD);
    actual_ops.end();
    Timer more_mem;
    more_mem.start();

    hipMemcpy(result.data(), d_result, n * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_input1);
    hipFree(d_input2);
    hipFree(d_result);
    more_mem.end();
    std::cout << "Time spent on fft ops: " << std::fixed << std::setprecision(10) << (actual_ops.get_duration<std::chrono::microseconds>() / 1e6) << '\n';
    std::cout << "Time spent on memory operations: " << std::fixed << std::setprecision(10) << (more_mem.get_duration<std::chrono::microseconds>() / 1e6 + memory_timer.get_duration<std::chrono::microseconds>() / 1e6 ) << '\n';
    return result;
}

std::vector<std::vector<std::complex<double>>> convertToComplex2D(
    const std::vector<uint32_t>& input, 
    uint32_t num_blocks, 
    uint32_t T) {

    std::vector<std::vector<std::complex<double>>> output(num_blocks, std::vector<std::complex<double>>(T));
    
    for (uint32_t i = 0; i < num_blocks; ++i) {
        for (uint32_t j = 0; j < T; ++j) {
            // Convert input[i * T + j] to std::complex<double>
            double realPart = static_cast<double>(input[i * T + j]);
            output[i][j] = std::complex<double>(realPart, 0.0); // imaginary part is 0
        }
    }

    return output;
}

bool solve_fft(const std::vector<uint32_t>& w, const uint32_t T){

    Timer init_timer;
    init_timer.start();
    const int n = std::size(w);
    uint32_t num_blocks = (n + NAIVE_SIZE - 1) / NAIVE_SIZE;
    //std::cout << "numblocks " << num_blocks << "\n";
    const int num_iterations = std::__lg(num_blocks);
    

    size_t global_dp_size = 2 * (T+1) * num_blocks * sizeof(uint32_t);
    uint32_t* global_dp;
    
    
    // blocks of size num_blocks*sum+1
    std::vector<uint32_t> blocks(num_blocks * (T + 1), 0);
    hipMalloc(&global_dp, global_dp_size);
    
    uint32_t* d_w;
    hipMalloc(&d_w, n * sizeof(uint32_t));
    hipMemcpy(d_w, w.data(), n * sizeof(uint32_t), hipMemcpyHostToDevice);

    

    uint32_t* d_blocks;
    hipMalloc(&d_blocks, num_blocks*(T+1)*sizeof(uint32_t));

    init_timer.end();
    Timer subsetkernel_timer;
    subsetkernel_timer.start();
    subsetSumKernelDp<<<num_blocks, BLOCKSIZE>>>(d_w, global_dp, n, T, d_blocks);
    subsetkernel_timer.end();
    Timer middleops_timer;
    middleops_timer.start();
    hipMemcpy(blocks.data(), d_blocks, num_blocks*(T+1) * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipFree(d_blocks);
    hipFree(global_dp);
    hipFree(d_w);
    
    // create an array of cuda complex doubles
    std::vector fft_outputs(num_iterations + 1, std::vector<std::vector<std::complex<double>>>(num_blocks));
    fft_outputs[0] = convertToComplex2D(blocks, num_blocks, T+1);
    //std::cout << fft_outputs[0][0][5];
    middleops_timer.end();
    Timer fft_merge_timer;
    fft_merge_timer.start();
    for (int iter = 0, iter_num_blocks = num_blocks; iter < num_iterations; iter++, iter_num_blocks = (iter_num_blocks + 1) / 2) {
        const int next_iter_num_blocks = (iter_num_blocks + 1) / 2;

        for (int i = 0; i < next_iter_num_blocks; i++) {
            if (2 * i + 1 < iter_num_blocks) {
                fft_outputs[iter + 1][i] = fftConvolutionCuFFT(fft_outputs[iter][2 * i], fft_outputs[iter][2 * i + 1]);

            } else {
                fft_outputs[iter + 1][i] = std::move(fft_outputs[iter][2 * i]);
            }
        }
    }
    fft_merge_timer.end();

    

    //std::cout << "Time spent combining subarray solutions: " << std::fixed << std::setprecision(10) << (convolution_timer.get_duration<std::chrono::microseconds>() / 1e6) << '\n';

    bool is_possible = (fft_outputs[num_iterations][0][T].real()>0);

    std::cout << "Time spent on fft_merge: " << std::fixed << std::setprecision(10) << (fft_merge_timer.get_duration<std::chrono::microseconds>() / 1e6) << '\n';
    std::cout << "Time spent on middle_ops: " << std::fixed << std::setprecision(10) << (middleops_timer.get_duration<std::chrono::microseconds>() / 1e6) << '\n';
    std::cout << "Time spent on subsetkernel: " << std::fixed << std::setprecision(10) << (subsetkernel_timer.get_duration<std::chrono::microseconds>() / 1e6) << '\n';
    std::cout << "Time spent on init: " << std::fixed << std::setprecision(10) << (init_timer.get_duration<std::chrono::microseconds>() / 1e6) << '\n';
    return is_possible;

}
