#include <hip/hip_runtime.h>
#include <cstdint>

#include <vector>
#include <iostream>
#include <iomanip>

__global__ void subsetSumKernelRow(uint32_t* dp_current, uint32_t* dp_previous, const uint32_t* set, int i, uint32_t sum) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j <= sum) {
        if (j >= set[i - 1]) {
            dp_current[j] = dp_previous[j] || dp_previous[j - set[i - 1]];
        } else {
            dp_current[j] = dp_previous[j];
        }
    }
}

bool subsetSumExists(const std::vector<uint32_t>&w, uint32_t T) {
    int n = w.size();

    std::vector<uint32_t> dp(2 * (T + 1), 0);
    dp[0] = 1;

    uint32_t *d_dp_current;
    uint32_t *d_dp_previous;
    hipMalloc(&d_dp_current, (T + 1) * sizeof(uint32_t));
    hipMalloc(&d_dp_previous, (T + 1) * sizeof(uint32_t));
    hipMemcpy(d_dp_previous, dp.data(), (T + 1) * sizeof(uint32_t), hipMemcpyHostToDevice);

    uint32_t* d_w;
    hipMalloc(&d_w, n * sizeof(uint32_t));
    hipMemcpy(d_w, w.data(), n * sizeof(uint32_t), hipMemcpyHostToDevice);


    dim3 dimBlock(256);
    dim3 dimGrid((T + 256 - 1) / 256);

    for (int i = 1; i <= n; ++i) {
        subsetSumKernelRow<<<dimGrid, dimBlock>>>(d_dp_current, d_dp_previous, d_w, i, T);
        hipDeviceSynchronize();

        std::swap(d_dp_current, d_dp_previous);
    }
    hipMemcpy(dp.data(), d_dp_previous, (T + 1) * sizeof(uint32_t), hipMemcpyDeviceToHost);

    bool result = dp[T];

    hipFree(d_dp_current);
    hipFree(d_dp_previous);
    hipFree(d_w);

    return result;
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}

